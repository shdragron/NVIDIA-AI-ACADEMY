#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  hipStream_t streamA,streamB,streamC,streamD,streamE;       // CUDA streams are of type `hipStream_t`.
  hipStreamCreate(&streamA);
  hipStreamCreate(&streamB);
  hipStreamCreate(&streamC);
  hipStreamCreate(&streamD);
  hipStreamCreate(&streamE);
    
  
  

  printNumber<<<1, 1, 0, streamA>>>(0);
  printNumber<<<1, 1, 0, streamB>>>(1);
  printNumber<<<1, 1, 0, streamC>>>(2);
  printNumber<<<1, 1, 0, streamD>>>(3);
  printNumber<<<1, 1, 0, streamE>>>(4);
    

    
  hipDeviceSynchronize();
  hipStreamDestroy(streamA);
  hipStreamDestroy(streamB);
  hipStreamDestroy(streamC);
  hipStreamDestroy(streamD);
  hipStreamDestroy(streamE);
  
  
}

!nvcc -o print-numbers 05-stream-intro/01-print-numbers.cu -run