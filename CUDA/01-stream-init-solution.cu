#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *host_a, *device_a ;
  float *host_b, *device_b ;
  float *host_c, *device_c ;
  
  

  hipHostMalloc(&host_a, size);
  hipHostMalloc(&host_b, size);
  hipHostMalloc(&host_c, size);
  
    
  hipMalloc(&device_a, size);
  hipMalloc(&device_b, size);
  hipMalloc(&device_c, size);

    

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */
  hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
  hipMemcpy(device_b, host_b, size, hipMemcpyHostToDevice);
  hipMemcpy(device_c, host_c, size, hipMemcpyHostToDevice);
    
    
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, device_a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, device_b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, device_c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(device_c, device_a, device_b, N);

  hipMemcpy(host_a, device_a, size, hipMemcpyDeviceToHost);
  hipMemcpy(host_b, device_b, size, hipMemcpyDeviceToHost);
  hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);
  
    
  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));


  checkElementsAre(7, host_c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);
}

!nvcc -o vector-add-manual-alloc 06-stream-init/solutions/01-stream-init-solution.cu -run